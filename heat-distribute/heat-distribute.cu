#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <time.h>
#include <mpi.h>
#include <hip/hip_runtime.h>

#define BUFF_LEN 256 
#define N 64
#define WIDTH 10000
#define HEIGH 10000
#define NUM_POINTS 2000
#define MAX_TEMP 10000
#define NUM_PROCS 2
#define NUM_LOOPS 2000
// Enumeration of CUDA devices accessible for the process.
void enumCudaDevices(char *buff)
{
    char tmpBuff[BUFF_LEN];
    int i, devCount;

    hipGetDeviceCount(&devCount);
    sprintf(tmpBuff," %3d", devCount);
    strncat(buff, tmpBuff, BUFF_LEN);

    for (i = 0; i < devCount; i++)
    {
        hipDeviceProp_t devProp;

        hipGetDeviceProperties(&devProp, i);
        sprintf(tmpBuff, "  %d:%s", i, devProp.name);
        strncat(buff, tmpBuff, BUFF_LEN);
    }
    
}

void test_dst(int* dev_dst, int num_rows)
{
	int* test_dst = (int* )malloc((num_rows)*WIDTH*sizeof(int));
	hipMemcpy(test_dst, dev_dst, (num_rows)*WIDTH*sizeof(int), hipMemcpyDeviceToHost);
	for(int i = 0; i<num_rows;i++)
	{
		for(int j = 0; j <WIDTH;j++)
			printf("%d ",test_dst[i*WIDTH+j]);
		printf("\n");
	}
}

void test_matrix(int* matrix, int num_rows)
{
	for(int i = 0; i<num_rows;i++)
	{
		for(int j = 0; j <WIDTH;j++)
			printf("%d ",matrix[i*WIDTH+j]);
		printf("\n");
	}
}

int* init_matrix ()
{
	srand(1234);
	printf("generating input matrix size %d X %d with %d random heat sources \n",WIDTH,HEIGH,NUM_POINTS);
	int* matrix = (int*) malloc((WIDTH*HEIGH)*sizeof(int));
	for(int i = 0; i < NUM_POINTS; i++)
	{
		int width_index = rand()%WIDTH;
		int heigh_index = rand()%HEIGH;
		matrix[WIDTH*heigh_index + width_index] = rand()%MAX_TEMP ;	
//		printf("width : %d    heigh : %d \n",width_index, heigh_index);
	}

	return matrix;
}

int* scatter_matrix(int* source_matrix, int rows_per_proc)
{
	int size = rows_per_proc*WIDTH*sizeof(int);
	int *per_proc_matrix = (int*) malloc(size);
	//MPI_Scatter(matrix, rows_per_proc*WIDTH , MPI_INT, recv_matrix, rows_per_proc*WIDTH , MPI_INT, 0 , MPI_COMM_WORLD);
	per_proc_matrix= (int*) malloc(rows_per_proc*WIDTH*sizeof(int));
	MPI_Scatter(source_matrix, rows_per_proc*WIDTH , MPI_INT, per_proc_matrix , rows_per_proc*WIDTH , MPI_INT, 0 , MPI_COMM_WORLD);

//	printf("\n \ntest scatter_src \n");	
//	test_matrix(per_proc_matrix, rows_per_proc);
	
	return per_proc_matrix;	
}

__global__ void heat_distribute(int* source_matrix, int* dest_matrix,int numthreads)
{

	int index = threadIdx.x + blockIdx.x * blockDim.x;
	if(index <numthreads)
	{
		int heigh_index = index/WIDTH + 1 ;
		int width_index = index%WIDTH;
		int sum=0;
		for(int x = -1; x <2; x++)
		 for(int y=-1; y < 2 ;y++)
		{
			int new_width_index = width_index + x;
			int new_heigh_index = heigh_index + y;
			sum += 1.0f*source_matrix[new_heigh_index*WIDTH  + new_width_index];
		}	
		dest_matrix[index+WIDTH] = sum/9;
	}
} 

void update_matrix (int* dev_src, int* dev_dst, int rows_per_proc)
{
	hipMemcpy(dev_src, dev_dst, (rows_per_proc+2)*WIDTH*sizeof(int), hipMemcpyDeviceToDevice);
}


int** init_gpu_memory(int*src_matrix,int rows_per_proc,int myrank)
{
	int* dev_src, *dev_dst;
	int row_size = WIDTH*sizeof(int);

	int size = rows_per_proc*row_size;

	hipMalloc((void**)&dev_src,(rows_per_proc+2)*row_size);
	hipMalloc((void**)&dev_dst,(rows_per_proc+2)*row_size);
	
	hipMemcpy(dev_src+WIDTH, src_matrix, size, hipMemcpyHostToDevice);
	hipMemcpy(dev_dst , dev_src, size+2*row_size, hipMemcpyDeviceToDevice);
	
	
	
	int ** result = (int**) malloc(2*sizeof(int*));
	result[0] = dev_src;
	result[1]= dev_dst;
	//~ printf("\n \ntest matrix_src %d \n",myrank);	
	//~ test_matrix(src_matrix, rows_per_proc);
	//~ printf("\n \ntest dev_src %d\n",myrank);	
	//~ test_dst(dev_src, rows_per_proc+2);
	//~ printf("\n \ntest dev_dst %d\n",myrank);
	//~ test_dst(dev_dst,rows_per_proc+2);
	
	return result;	
}

void exchange_data(int* dev_src, int rows_per_proc, int myrank, int numprocs)
{

	int row_size = WIDTH*sizeof(int);
	int* send_row = (int*)malloc(row_size);
	int *recv_row = (int*) malloc (row_size);
	MPI_Status* status = new MPI_Status();

	//int offset = (rows_per_proc+1) * row_size;
	
	if(myrank == 0)
	{
		int offset = (rows_per_proc) * WIDTH;
		hipMemcpy(send_row, dev_src+offset, row_size, hipMemcpyDeviceToHost);
		
	
		MPI_Send(send_row, WIDTH , MPI_INT, myrank+1, 0, MPI_COMM_WORLD);
		MPI_Recv(recv_row, WIDTH , MPI_INT, myrank+1 ,0, MPI_COMM_WORLD,status);
		
		hipMemcpy(dev_src+offset+WIDTH, recv_row, row_size, hipMemcpyHostToDevice);
	}	
	if (myrank == numprocs-1)
	{
		int offset = WIDTH;
		hipMemcpy(send_row, dev_src+offset, row_size, hipMemcpyDeviceToHost);
		//~ printf("test send row %d :\n",myrank);
		//~ for(int i=0;i<WIDTH;i++)
			//~ printf("%d ",send_row[i]);
		//~ printf("\n");
		//~ 
		MPI_Send(send_row, WIDTH , MPI_INT, myrank-1, 0, MPI_COMM_WORLD);
		MPI_Recv(recv_row, WIDTH , MPI_INT, myrank-1 ,0, MPI_COMM_WORLD,status);
		
		hipMemcpy(dev_src, recv_row, row_size, hipMemcpyHostToDevice);		
	}
	else
	{
	//TODO: more than two GPUs 
	}
	
	//~ printf("\n \ntest exchange data %d\n",myrank);	
	//~ test_dst(dev_src, rows_per_proc+2);

}

void run_heat_kernel(int myrank, int* dev_src, int* dev_dst,int numprocs, int rows_per_proc)
{

	//~ printf("row per proc: %d \n", rows_per_proc);
	int numthreads = rows_per_proc*WIDTH;
	if(numprocs >1)
		exchange_data(dev_src, rows_per_proc, myrank, numprocs);	
	
	heat_distribute<<<1,numthreads>>>(dev_src,dev_dst,numthreads);

	//~ printf("\n \ntest dev_src %d\n",myrank);	
	//~ test_dst(dev_src, rows_per_proc+2);
	//~ printf("\n \ntest dev_dst %d\n",myrank);
	//~ test_dst(dev_dst,rows_per_proc+2);
	MPI_Barrier(MPI_COMM_WORLD);
	
	update_matrix (dev_src, dev_dst, rows_per_proc);
	

   	//printf ("return from kernel %d , with value = %d \n", myrank, c);
	
}


int main(int argc, char *argv[])
{
    int i, myrank, numprocs;
    char pName[MPI_MAX_PROCESSOR_NAME],
    buff[BUFF_LEN];

	int* source_matrix;
	int* per_proc_matrix;
	int* dev_src;
	int* dev_dst;
	int rows_per_proc;
	
		double start_time;
		double end_time; 
    MPI_Init(&argc,&argv);
    MPI_Comm_size(MPI_COMM_WORLD,&numprocs);
    MPI_Comm_rank(MPI_COMM_WORLD, &myrank);
    MPI_Get_processor_name(pName, &i);	
	
	rows_per_proc = HEIGH/numprocs;

    sprintf(buff, "%-15s %3d", pName, myrank);

// Find local CUDA devices

    enumCudaDevices(buff);
//    run_add_kernel(myrank);
    
// Collect and print the list of CUDA devices from all MPI processes
    if (myrank == 0)
    {
        char devList[10][BUFF_LEN];
 
        MPI_Gather(buff, BUFF_LEN, MPI_CHAR,devList, BUFF_LEN, MPI_CHAR,0, MPI_COMM_WORLD);
        for (i = 0; i < numprocs; i++)
            printf("%s\n", devList + i);
    }
    else
        MPI_Gather(buff, BUFF_LEN, MPI_CHAR, NULL, 0, MPI_CHAR, 0, MPI_COMM_WORLD);

//heat distribution begin here


   if(myrank == 0)
	{
		source_matrix = init_matrix();
		//~ for(int i=0; i<HEIGH ; i++)
		//~ {
			//~ for(int j = 0; j<WIDTH;j++)
				//~ printf("%d ", source_matrix[i*WIDTH+j]);
			//~ printf("\n");
		//~ }
	printf("rows per proccess: %d \n", rows_per_proc);
	
	printf("launching computing kernel .... \n\n");
	start_time = MPI_Wtime();
	}



	if(numprocs>1)
	{
		per_proc_matrix = scatter_matrix(source_matrix,rows_per_proc);
	
	//	per_proc_matrix= (int*) malloc(rows_per_proc*WIDTH*sizeof(int));
	//	MPI_Scatter(source_matrix, rows_per_proc*WIDTH , MPI_INT, per_proc_matrix , rows_per_proc*WIDTH , MPI_INT, 0 , MPI_COMM_WORLD);
	
		//~ printf("\n \ntest scatter_src %d \n", myrank);	
		//~ test_matrix(per_proc_matrix, rows_per_proc);
	}
	else
	{
		per_proc_matrix = (int *) malloc(WIDTH*HEIGH*sizeof(int));
		for(int i =0;i <WIDTH*HEIGH; i++)
			per_proc_matrix[i] = source_matrix[i];	
	}
	
	
	int ** dev_pointer = init_gpu_memory(per_proc_matrix, rows_per_proc, myrank);	
	dev_src = dev_pointer[0];
	dev_dst = dev_pointer[1];
	
	

	for(int i = 0; i<NUM_LOOPS; i++)
	run_heat_kernel(myrank, dev_src, dev_dst, numprocs, rows_per_proc);


	hipMemcpy(per_proc_matrix, dev_dst+WIDTH, rows_per_proc*WIDTH*sizeof(int), hipMemcpyDeviceToHost);
	
	
	if(numprocs >1)
		MPI_Gather(per_proc_matrix, rows_per_proc*WIDTH, MPI_INT, source_matrix ,rows_per_proc*WIDTH, MPI_INT, 0 , MPI_COMM_WORLD);
	else
	{
		for(int i =0 ; i < WIDTH*HEIGH; i++)
			source_matrix[i] = per_proc_matrix[i];
	}
	
	if(myrank == 0)
	 {
		 end_time = MPI_Wtime();
		 printf("Elapsed time: %f secs", (end_time - start_time));
		//~ printf("\n result : \n");
		//~ for(int i=0; i<HEIGH ; i++)
		//~ {
			//~ for(int j = 0; j<WIDTH;j++)
				//~ printf("%d ", source_matrix[i*WIDTH+j]);
			//~ printf("\n");
		//~ }
	}

	 MPI_Finalize();
    return 0;
}


